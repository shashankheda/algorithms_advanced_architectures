#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define NUM_THREAD 512
#define NUM_BLOCK 65

// Function to sum an array
__global__ void reduce0(float *g_odata) {
extern __shared__ int sdata[];

// each thread loads one element from global to shared mem
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
sdata[tid] = g_odata[i];
__syncthreads();

// do reduction in shared mem
for (unsigned int s=1; s < blockDim.x; s *= 2) { // step = s x 2
    if (tid % (2*s) == 0) { // only threadIDs divisible by the step participate
        sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
}

// write result for this block to global mem
if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

///////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////
__global__ void monteCarlo(float *g_odata, int  trials, hiprandState *states){
    extern __shared__ int sdata[];
//  unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int k, incircle;
    float x, y, z;
    incircle = 0;

    hiprand_init(1234, i, 0, &states[i]);

    for(k = 0; k < trials; k++){

    x = hiprand_uniform(&states[i]);
    y = hiprand_uniform(&states[i]);
    z = sqrt(x*x + y*y);
    if (z <= 1) incircle++;
    else{}
    }
    __syncthreads();
    g_odata[i] = incircle;
}
///////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////
int main() {

    float* solution = (float*)calloc(100, sizeof(float));
    float *sumDev, sumHost[NUM_BLOCK*NUM_THREAD];
    int trials, total; 
    hiprandState *devStates;



    trials = 100;
    total = trials*NUM_THREAD*NUM_BLOCK;

    dim3 dimGrid(NUM_BLOCK,1,1); // Grid dimensions
    dim3 dimBlock(NUM_THREAD,1,1); // Block dimensions
    size_t size = NUM_BLOCK*NUM_THREAD*sizeof(float); //Array memory size
    hipMalloc((void **) &sumDev, size); // Allocate array on device

//    cudaMalloc((void **) &devStates, size*sizeof(curandState));
hipMalloc((void **) &devStates, (NUM_BLOCK *NUM_THREAD)*sizeof(hiprandState));
    // Do calculation on device by calling CUDA kernel
    monteCarlo <<<dimGrid, dimBlock, size>>> (sumDev, trials, devStates);
        // call reduction function to sum
    reduce0 <<<dimGrid, dimBlock, size>>> (sumDev);
    // Retrieve result from device and store it in host array
    hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);

    *solution = 4*(sumHost[0]/total);
    printf("\n%.*f\n", 1000, *solution);
    free (solution);
    //*solution = NULL;
    return 0;
}